#include "hip/hip_runtime.h"
#include <math.h>

#include "calculate_normq_cu_v1.h"

__device__ void calculate_normq_cu_c( double *normq, double *dfmlc, double *v, int c, int n )
{
	int f;

	normq[c] = 0.0;

    for ( f = 0; f < n; ++f )
	{
   	    normq[c] += ( ( v[f] * v[f] ) / ( dfmlc[ f + n * c ] * dfmlc[ f + n * c ] ) );
    }

    normq[c]  = sqrt( normq[c] );
}

__global__ void calculate_normq_cu( double *normq, double *dfmlc, double *v, int n )
{
	int c;

	c = threadIdx.x + blockDim.x * blockIdx.x;

	while ( c < n ) //for ( c = 0; c < n; ++c )
    {
		calculate_normq_cu_c( normq, dfmlc, v, c, n );

		c += blockDim.x * gridDim.x;
	}
}

