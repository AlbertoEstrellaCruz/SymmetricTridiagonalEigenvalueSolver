#include "hip/hip_runtime.h"
#include <math.h>

#include "calculate_vi_cu_v1.h"

__device__ void calculate_vi_cu( double *pvi, double *d, double *dfmlc, double *v, int i, int n )
{
	int j;

	double vi;

	vi = dfmlc[ i + n * i ];

	for ( j = 0; j < i ; ++j )
	{
    	vi *= ( dfmlc[ i + n * j ] / ( d[j] - d[i] ) );
	}

	for ( j = i + 1; j < n; ++j )
	{
    	vi *= ( dfmlc[ i + n * j ] / ( d[j] - d[i] ) );
	}

	if ( n % 2 == 1 )
	{
    	vi = -vi;
	}
/*
	if ( vi < 0.0 )
	{
    	printf( "i = %d\n", i );
    	printf( "vi = %.15e\n", vi );
	}
*/
	vi = sqrt(vi);

	if ( v[i] < 0.0 )
	{
    	vi = -vi;
	}

	*pvi = vi;
}

#ifdef __cplusplus
extern "C" 
#endif
__global__ void correct_v_cu( double *v_corr, double *d, double *dfmlc, double *v, int n)
{
	int i;

	i = threadIdx.x + blockDim.x * blockIdx.x;

	while ( i < n ) //for ( i = 0; i < n; ++i )
	{
		calculate_vi_cu( v_corr + i, d, dfmlc, v, i, n );

		i += blockDim.x * gridDim.x;
	}
}

