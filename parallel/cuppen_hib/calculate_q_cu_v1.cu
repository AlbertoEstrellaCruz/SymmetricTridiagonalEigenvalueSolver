#include "hip/hip_runtime.h"
#include "calculate_q_cu_v1.h"

__device__ void calculate_q_cu_fc( double *q, double *v, double *dfmlc,  double *normq, int f, int c, int n )
{
	q[ f + n * c ] = ( v[f] / dfmlc[ f + n * c ] ) / normq[c];
}

__global__ void calculate_q_cu( double *q, double *v, double *dfmlc, double *normq, int n )
{
	int c;
	int f;

	c = threadIdx.y + blockDim.y * blockIdx.y;

	while ( c < n ) //for ( c = 0; c < n; ++c )
    {
		f = threadIdx.x + blockDim.x * blockIdx.x;

    	while ( f < n ) //for ( f = 0; f < n; ++f)
     	{
    	    calculate_q_cu_fc( q, v, dfmlc,  normq, f, c, n );

			f += blockDim.x * gridDim.x;
    	}

		c += blockDim.y * gridDim.y;
	}
}

