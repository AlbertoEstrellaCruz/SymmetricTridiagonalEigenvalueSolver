#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "zerodandc_n_cu_v1.h"
#include "calculate_vi_cu_v1.h"
#include "calculate_normq_cu_v1.h"
#include "calculate_q_cu_v1.h"

#include "rank_one_cu_v1.h"

__device__ void calculate_v2_cu_r( double *v2, double *v, int r )
{
	v2[r] = v[r] * v[r];
}

__global__ void calculate_v2_cu( double *v2, double *v, int n )
{
	int r;

	r = threadIdx.x + blockDim.x * blockIdx.x;

	while ( r < n ) //for ( r = 0; r < n; ++r )
	{
		calculate_v2_cu_r( v2, v, r );

		r += blockDim.x * gridDim.x;
	}
}

#ifdef __cplusplus
extern "C" 
#endif
void rank_one_cuda( double *q, double *lambda, double *d, double *v, int n, double eps )
{
	int n_size;
	int nxn_size;

	double *q_d;
	double *lambda_d;
	double *d_d;
	double *v_d;

	double *delta_d;
	double *dfmlc_d;
	double *v2_d;
	double *v_corr_d;
	double *normq_d;

	dim3 n_grid_dim;
	dim3 n_block_dim;
	dim3 nxn_grid_dim;
	dim3 nxn_block_dim;

	n_size = n * sizeof( double );
	nxn_size = n * n_size;

	n_block_dim = 32;
	n_grid_dim = n / 32 + 1;

	nxn_block_dim.x = n_block_dim.x;
	nxn_block_dim.y = n_block_dim.x;
	nxn_grid_dim.x = n_grid_dim.x;
	nxn_grid_dim.y = n_grid_dim.x;

	hipMalloc( &q_d, nxn_size );

	hipMalloc( &delta_d, nxn_size );
	hipMalloc( &dfmlc_d, nxn_size );

	hipMalloc( &lambda_d, n_size );
	hipMalloc( &d_d, n_size );
	hipMalloc( &v_d, n_size );

	hipMalloc( &v2_d, n_size );
	hipMalloc( &v_corr_d, n_size );
	hipMalloc( &normq_d, n_size );

	hipMemcpy( v_d, v, n_size, hipMemcpyHostToDevice );
	hipMemcpy( d_d, d, n_size, hipMemcpyHostToDevice );

  // Calculando v al cuadrado

	calculate_v2_cu<<< n_grid_dim, n_block_dim >>>( v2_d, v_d, n );

  // Calculando los valores propios de la ecuacion secular

	solve_secular_cu<<< n_grid_dim, n_block_dim >>>( delta_d, lambda_d, dfmlc_d, d_d, v2_d, n, eps );

  // Calculando la correccion de v 

	correct_v_cu<<< n_grid_dim, n_block_dim >>>( v_corr_d, d_d, dfmlc_d, v_d, n );

  // Calculando los vectores propios en Q

	calculate_normq_cu<<< n_grid_dim, n_block_dim >>>( normq_d, dfmlc_d, v_corr_d, n);

	calculate_q_cu<<< nxn_grid_dim, nxn_block_dim >>>( q_d, v_corr_d, dfmlc_d, normq_d, n );

	hipMemcpy( lambda, lambda_d, n_size, hipMemcpyDeviceToHost );
	hipMemcpy( q, q_d, nxn_size, hipMemcpyDeviceToHost );

	hipFree( normq_d );
	hipFree( v_corr_d );
	hipFree( v2_d );
	hipFree( v_d );
	hipFree( d_d );
	hipFree( lambda_d );
	hipFree( dfmlc_d );
	hipFree( delta_d );
	hipFree( q_d );
}

