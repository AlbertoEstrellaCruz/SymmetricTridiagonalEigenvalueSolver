#include <stdlib.h>
#include <stdio.h>

#include "hipblas.h"

#include "mult_cu_v1.h"

#ifdef __cplusplus
extern "C" 
#endif
void mult_cuda( double *c , double *a , double *b, int m, int k, int n )
{
	hipblasStatus_t stat;
	hipblasHandle_t handle;

	double alpha;
	double beta;

	double *d_c;
	double *d_a;
	double *d_b;

	alpha = 1.0;
	beta = 0.0;

	hipMalloc( &d_a, m * k * sizeof( double ) );
	hipMalloc( &d_b, k * n * sizeof( double ) );
	hipMalloc( &d_c, m * n * sizeof( double ) );

	hipMemcpy( d_a, a, m * k * sizeof( double ), hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, k * n * sizeof( double ), hipMemcpyHostToDevice );

	stat = hipblasCreate( &handle );

	if ( stat != HIPBLAS_STATUS_SUCCESS ) {
        fprintf ( stderr, "CUBLAS initialization failed\n" );
		return;
    }

	stat = hipblasDgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_a, m, d_b, k, &beta, d_c, m );

	if ( stat != HIPBLAS_STATUS_SUCCESS ) {
        fprintf ( stderr, "CUBLAS hipblasDgemm failed\n" );
		return;
    }

	hipMemcpy( c, d_c, m * n * sizeof( double ), hipMemcpyDeviceToHost );

	hipFree( d_c );
	hipFree( d_b );
	hipFree( d_a );

	hipblasDestroy(handle);
}

